#include "hip/hip_runtime.h"
//puissance_cuda.c

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

#include "defs.h"

#define TAILLE_BLOC_X 256
#define NB_ELEM 16
#define T 16

//nb de bloc : gridDim.x
//indice de parcours des blocs : blockIdx.x

//nb de thread par bloc : blockDim.x
//indice de parcours des threads : threadIdx.x

__global__ void matmulKernel(REAL_T* d_A, REAL_T* d_B, REAL_T* d_C, int n,REAL_T *norm) {

   int ib = blockIdx.y;
   int jb = blockIdx.x;

   int i = threadIdx.y;
   int j = threadIdx.x;

    float tmp = 0;

    // Mémoire partagée pour stocker les sous-blocs de A et B :
   __shared__ float As[T][T];
   __shared__ float Bs[T];
   if(j==0)
   {
     float *p_d_B = d_B + (T*ib);
     Bs[i] = p_d_B[i];
   }

   // Boucle sur les sous-blocs de A et B
   // Adresse du premier élément du sous-bloc A(ib, m) et du sous-bloc B(m, jb) :
   float *p_d_A = d_A + (n*T*ib + T*jb);
   // Chaque thread charge un élément de A et de B (globale → partagée) :
   As[i][j] = p_d_A[n * i + j];
   // Synchronisation avant calcul nécessaire entre les threads du bloc :
   __syncthreads();

   // Multiplication des sous-blocs A(ib, m) et B(m, jb) :
   for (int e=0; e<T; ++e)
   tmp += As[i][e] * Bs[e];
   // Synchronisation pour garantir que le calcul est terminé
   // pour tous les threads du bloc avant de ré-écrire dans As et Bs
   // (à l'itération suivante) :
   __syncthreads();
   // Chaque thread écrit son élément de C en mémoire globale :
   atomicAdd(&d_C[i],tmp);
}

__global__ void norm_to_Kernel(REAL_T* d_C, REAL_T *norm) {
  unsigned int i = blockDim.x*blockIdx.x+threadIdx.x;
  REAL_T temp;
  REAL_T inter;
  for(int line = 0; line<NB_ELEM; line++)
  {
    int line_n = i*NB_ELEM+line;
    temp = 0;
    temp = d_C[line_n];
    inter = temp*temp;
    atomicAdd(norm,inter);
  }
}


__global__ void normKernel(REAL_T* d_Y,REAL_T* d_X, REAL_T *norm, int n, REAL_T* erreur) {
  unsigned int i = blockDim.x*blockIdx.x+threadIdx.x;

  if (i*NB_ELEM < n)
  {
    REAL_T temp;
    REAL_T inter;
    for(int line = 0; line<NB_ELEM; line++)
    {
      int line_n = i*NB_ELEM+line;
      temp = d_Y[line_n]/(*norm);
      d_Y[line_n] = temp;
      inter =  d_X[line_n] - temp;
      inter = inter*inter;
      atomicAdd(erreur,inter);
    }
  }
}

__global__ void errorKernel(REAL_T *erreur) {
  *erreur = sqrt(*erreur);
}


//Création de la matrice sur CPU pour accès GPU
int main(int argc, char **argv){

    long i, n;
    long long size;
    REAL_T *norm, *error, error_cpu,norm_cpu,zero=0;
    error_cpu = 99;
    REAL_T *A, *A_i, *X, *Y, *d_A, *d_X,*d_Y;
    double start_time, total_time;
    int n_iterations;
    FILE *output;



    if (argc < 2){
        printf("USAGE: %s [n]\n", argv[0]);
        exit(1);
    }
    n = atoll(argv[1]);
    size = n * n * sizeof(REAL_T);
    printf("taille de la matrice : %.1f G\n", size / 1073741824.);

    /*** allocation de la matrice et des vecteurs ***/
    A = (REAL_T *)malloc(size);
    if (A == NULL) {
        perror("impossible d'allouer la matrice");
        exit(1);
    }
    X = (REAL_T *)malloc(n * sizeof(REAL_T));
    Y = (REAL_T *)malloc(n * sizeof(REAL_T));
    if ((X == NULL) || (Y == NULL)) {
        perror("impossible d'allouer les vecteur");
        exit(1);
    }

    /*** initialisation de la matrice et de x ***/
    A_i = A;
    for (i = 0; i < n; i++) {
        init_ligne(A_i, i, n);
        A_i += n;
    }

    for (i = 0; i < n; i++) {
        X[i] = 1.0 / n;
    }

    //Initialisation de variables
    start_time = my_gettimeofday();
    n_iterations = 0;

    // Allocation GPU
    hipMalloc((void **) &d_A, size);
    hipMalloc((void **) &d_X, n * sizeof(REAL_T));
    hipMalloc((void **) &d_Y, n * sizeof(REAL_T));
    hipMalloc((void **) &error, 1 * sizeof(REAL_T));
    hipMalloc((void **) &norm, 1 * sizeof(REAL_T));

    //Transfert CPU → GPU
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_X, X, n * sizeof(REAL_T), hipMemcpyHostToDevice);

    dim3 threadsParBloc2D(T, T);
    dim3 tailleGrille2D(ceil(n/(float) T),ceil(n/(float) T));

    dim3 threadsParBloc(ceil(TAILLE_BLOC_X/(float)NB_ELEM));
    dim3 tailleGrille(ceil(n/(float) TAILLE_BLOC_X));

    dim3 threadsParBloc2(1);
    dim3 tailleGrille2(1);

    //BOUCLE DE CALCUL
    while (error_cpu > 0.00005) {

      //KERNEL 1 : Multipication matrice
      //Lancement de kernel (asynchrone) :
      //Définition des variables GPU
      hipMemcpy(norm, &zero, 1*sizeof(REAL_T), hipMemcpyHostToDevice);
      hipMemcpy(error, &zero, 1*sizeof(REAL_T), hipMemcpyHostToDevice);

      matmulKernel<<< tailleGrille2D, threadsParBloc2D>>>(d_A, d_X, d_Y, n, norm);
      //KERNEL 2 : Norme euclidienne total
      //Définition des variables GPU
      norm_to_Kernel<<< tailleGrille, threadsParBloc>>>(d_Y,norm);

      hipMemcpy(&norm_cpu, norm, 1*sizeof(REAL_T), hipMemcpyDeviceToHost);
      norm_cpu = sqrt(norm_cpu);
      hipMemcpy(norm, &norm_cpu, 1*sizeof(REAL_T), hipMemcpyHostToDevice);
      printf("Norm: %g\n",norm_cpu);
      //KERNEL 3 : Applique la norme
      normKernel<<< tailleGrille, threadsParBloc>>>(d_Y, d_X, norm, n, error);

      // KERNEL 4 : Ecart quadratique
      errorKernel<<< threadsParBloc2, tailleGrille2>>>(error);

      //COMMUNICATION GPU -> CPU
      hipMemcpy(&error_cpu, error, 1*sizeof(REAL_T), hipMemcpyDeviceToHost);
      printf("Err: %g\n",error_cpu);
      hipMemcpy(X, d_Y, n * sizeof(REAL_T), hipMemcpyDeviceToHost);
      hipMemcpy(d_X, X, n * sizeof(REAL_T), hipMemcpyHostToDevice);

      n_iterations ++ ;

    }
    hipMemcpy(Y, d_Y, n, hipMemcpyDeviceToHost);
    hipMemcpy(&norm_cpu, norm, 1*sizeof(REAL_T), hipMemcpyDeviceToHost);

    total_time = my_gettimeofday() - start_time;

    printf("Nombre d'op/thread : %d, Nombre de threads: %d, Nombre de blocs: %d\n",n*NB_ELEM,threadsParBloc.x,tailleGrille.x);
    printf("erreur finale après %4d iterations : %g (|VP| = %g)\n", n_iterations, error_cpu, norm_cpu);
    printf("temps : %.1f s      Mflop/s : %.1f \n", total_time, (2.0 * n * n + 7.0 * n) * n_iterations / 1048576. / total_time);
    /*** stocke le vecteur propre dans un fichier ***/
    output = fopen("result.out", "w");
    if (output == NULL) {
        perror("impossible d'ouvrir result.out en écriture");
        exit(1);
    }
    fprintf(output, "%ld\n", n);
    for (i = 0; i < n; i++) {
        fprintf(output, "%.17g\n", Y[i]);
    }
    fclose(output);

    /* Libération mémoire GPU et CPU : */

    hipFree(d_A); hipFree(d_X); hipFree(d_Y);
    free(A); free(X); free(Y);

  }
